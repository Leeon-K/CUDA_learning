#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

//2.386ms
__global__ void histgram(int *hist_data, int *bin_data)
{
    int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    /*error*/  
    // 内存访问冲突,
    // bin_data[hist_data[gtid]]++;
    /*right 要避免竞争*/
    atomicAdd(&bin_data[hist_data[gtid]], 1);
}

bool CheckResult(int *out, int* groudtruth, int N){
    for (int i = 0; i < N; i++){
        if (out[i] != groudtruth[i]) {
            return false;
        }
    }
    return true;
}

int main(){
    float milliseconds = 0;
    const int N = 25600000;
    int *hist = (int *)malloc(N * sizeof(int));
    int *bin = (int *)malloc(256 * sizeof(int));
    int *bin_data;
    int *hist_data;
    hipMalloc((void **)&bin_data, 256 * sizeof(int));
    hipMalloc((void **)&hist_data, N * sizeof(int));

    for(int i = 0; i < N; i++){
        hist[i] = i % 256;
    }

    int *groudtruth = (int *)malloc(256 * sizeof(int));;
    for(int j = 0; j < 256; j++){
        groudtruth[j] = 100000;
    }

    hipMemcpy(hist_data, hist, N * sizeof(int), hipMemcpyHostToDevice);
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    dim3 Grid(GridSize);
    dim3 Block(blockSize);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    histgram<<<Grid, Block>>>(hist_data, bin_data);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(bin, bin_data, 256 * sizeof(int), hipMemcpyDeviceToHost);
    bool is_right = CheckResult(bin, groudtruth, 256);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0; i < 256; i++){
            printf("%lf ", bin[i]);
        }
        printf("\n");
    }
    printf("histogram latency = %f ms\n", milliseconds);    

    hipFree(bin_data);
    hipFree(hist_data);
    free(bin);
    free(hist);
}