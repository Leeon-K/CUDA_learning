#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "hip/hip_runtime.h"
#include <algorithm>
template <typename T, int Size>
struct alignas(sizeof(T) * Size) AlignedVector {
  T val[Size];
  __host__ __device__ inline const T& operator[](int i) const { return val[i]; }
  __host__ __device__ inline T& operator[](int i) { return val[i]; }
};

__device__ float TanhApprox(float x) {
  //float r;
  //asm("tanh.approx.f32 %0,%1; \n\t" : "=f"(r) : "f"(x));
  //return r;
  return tanhf(x);
}

template<typename T>
struct GeluFunctor {
  static constexpr T alpha = static_cast<T>(0.7978845608028654);
  static constexpr T beta = static_cast<T>(0.044714998453855515);

  __device__ GeluFunctor() {};

  __device__ T operator()(T x) const {
    const T half = static_cast<T>(0.5);
    const T one = static_cast<T>(1);
    const T tanh_in = alpha * (x + beta * x * x * x);
    return half * x * (one + tanh(tanh_in));
  }
};

template<>
struct GeluFunctor<half> {
  static constexpr float alpha = GeluFunctor<float>::alpha;
  static constexpr float beta = GeluFunctor<float>::beta;
  GeluFunctor<float> float_functor;

  __device__ GeluFunctor() {};

  __device__ half operator()(const half x) const {
    // Note: when you have ampere GPU, you can enable the line45-50 method to get performance improvement by half intrinsic instead of static_cast half to fp32.
    const float tanh_in =
       __half2float(__float2half_rn(alpha) * (x + __float2half_rn(beta) * x * x * x));
    const float tanh_out = TanhApprox(tanh_in);
    return __float2half_rn(0.5f) * x * (__float2half_rn(1.0f) + __float2half_rn(tanh_out));
    // Note: half to float will lose performance using static_cast, because static_cast will be compiled to more instructions than half intrinsic,
    // so you should better use half intrinsic when you have ampere GPU, you can enable 44-47 line
    return static_cast<half>(float_functor(static_cast<float>(x)));
  }
//   Note: when you have ampere GPU, you can enable the "apply2" method to get performance improvement by half2 intrinsic.
  __device__ void Apply2(half* y, const half* x) const {
    const half2 x2 = *(reinterpret_cast<const half2*>(x));
    const float2 tanh_in = __half22float2(
       __hmul2(__float2half2_rn(alpha),
               __hadd2(x2, __hmul2(__hmul2(__hmul2(__float2half2_rn(beta), x2), x2), x2))));
    float2 tanh_out;
    tanh_out.x = TanhApprox(tanh_in.x);
    tanh_out.y = TanhApprox(tanh_in.y);
    const half2 y2 = __hmul2(__hmul2(__hmul2(__float2half2_rn(0.5F), x2),
                                    __hadd2(__float2half2_rn(1.0F), __float22half2_rn(tanh_out))));
    *reinterpret_cast<half2*>(y) = y2;
  }
};


template <int VecSize>
__global__ void FP16GeluCUDAKernel(const __half* x,
                                                 __half* y,
                                                 int n) {
  int offset =
      static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x) * VecSize;
  int stride = static_cast<int>(blockDim.x * gridDim.x) * VecSize;
  GeluFunctor<half> gelu_fwd;
  __half y_reg[VecSize];
  for (; offset < n; offset += stride) {
    using ArrT = AlignedVector<__half, VecSize>;
    const ArrT* in_arr = reinterpret_cast<const ArrT*>(x + offset);
    // ArrT* out_arr = reinterpret_cast<const ArrT*>(y + offset);
    const __half* in = reinterpret_cast<const __half*>(in_arr);
    // __half* out = reinterpret_cast<const __half*>(out_arr);

    if (VecSize == 1){
        y_reg[0] = gelu_fwd(in[0]);
    } else {
      // Note: when you have ampere GPU, you can enable the "apply2" method to get performance improvement by half2 intrinsic.
      //for (int i = 0; i < VecSize; i+=2) {
      //gelu_fwd.apply2(y + offset, in[i]);
        for (int i = 0; i < VecSize; i++) {
            y_reg[i] = gelu_fwd(in[i]);
        }
    }
    *reinterpret_cast<ArrT*>(y + offset) = *reinterpret_cast<ArrT*>(y_reg);
  }
}

int main() {
    int n = 1000;
    
    __half *x = new __half[n];
    __half *y = new __half[n];
    for (int i = 0; i < n; i++)
    {
      x[i] = (__half)(i);
    }
    __half * d_x, *d_y;
    hipMalloc((void **)&d_x, n * sizeof(__half));
    hipMalloc((void **)&d_y, n * sizeof(__half));
    hipMemcpy(d_x, x, sizeof(__half) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(__half) * n, hipMemcpyHostToDevice);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    auto is_aligned = [](const void* p, int alignment) {
        return reinterpret_cast<uintptr_t>(p) % alignment == 0;
    };
                                                                      
    constexpr auto kAlignment = alignof(AlignedVector<__half, 8>); 
    // Note: when you have ampere GPU, you can enable the 122-124 line to get performance improvement by half2 intrinsic.
    if (n % 8 == 0 && is_aligned(x, kAlignment) && is_aligned(y, kAlignment)) {                                          
      int thread = std::min<int>(512, deviceProp.maxThreadsPerBlock); 
      //int block = (n / 8 + thread - 1) / thread;                  
      //block = std::min<int>(block, deviceProp.maxGridSize[0]);                                  
      //FP16GeluCUDAKernel<8, true><<<block, thread>>>(x, y, n);  
      int block = (n + thread - 1) / thread;                  
      block = std::min<int>(block, deviceProp.maxGridSize[0]);                                  
      FP16GeluCUDAKernel<1><<<block, thread>>>(d_x, d_y, n);                      
      hipMemcpy(y, d_y, sizeof(__half) * n, hipMemcpyDeviceToHost);                                                          
    } 
    printf("pass");
    delete x;
    x = nullptr;
    delete y;
    y = nullptr;
    hipFree(d_x);
    hipFree(d_y);
}
