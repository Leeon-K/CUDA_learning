#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

// bug1:长时间运行无结果
// bug2:从某个index开始，cpu和gpu都为0
// bug1 fix: 检查kernel书写过程中是否发生低级错误，此处是line15写成了blockIdx.x*blockDim.x导致
// bug2 fix: cudaMemcpy/cudaMemcpyAsync的第三个参数是内存大小，而不是数据量，此处是写成数据量了
typedef float FLOAT;

/* CUDA kernel function */
__global__ void vec_add(FLOAT *x, FLOAT *y, FLOAT *z, int N)
{
    /* 1D grid */
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = idx; i < N; i += gridDim.x * blockDim.x){
    	z[i] = y[i] + x[i];
        //debug info
	//if(i==500) printf("index500,gpuz=%f,y=%f,x=%f\n",z[i],y[i],x[i]);
    }
}

void vec_add_cpu(FLOAT *x, FLOAT *y, FLOAT *z, int N)
{
    for (int i = 0; i < N; i++) {
        z[i] = y[i] + x[i];
        // same above
	//if(i==500) printf("i=500,z=%f\n",z[i]);
    }
}

int main()
{
    int N = 10000;
    int nbytes = N * sizeof(FLOAT);
    const int nstreams = 4;
    int nums_per_stream = N / nstreams;// assert N can be exactly divided by nstream
    int size_per_stream = nums_per_stream * sizeof(FLOAT); //就是少写了这里！！产生了bug2

    /* 1D block */
    int bs = 256;

    /* 1D grid */
    int s = ceil((size_per_stream + bs - 1.) / bs);
    dim3 grid(s);

    FLOAT *dx, *hx;
    FLOAT *dy, *hy;
    FLOAT *dz, *hz;

    /* allocate GPU mem */
    hipMalloc((void **)&dx, nbytes);
    hipMalloc((void **)&dy, nbytes);
    hipMalloc((void **)&dz, nbytes);
    
    /* init time */
    float milliseconds = 0;

    /* !must alllocate CPU pinned mem using cudaMallocHost*/
    hipHostAlloc(&hx, nbytes, hipHostMallocDefault);
    hipHostAlloc(&hy, nbytes, hipHostMallocDefault);
    hipHostAlloc(&hz, nbytes, hipHostMallocDefault);

    /* init */
    for (int i = 0; i < N; i++) {
        hx[i] = 1.0;
        hy[i] = 1.0;
    }
    hipStream_t streams[nstreams];
    
    for (int i = 0; i < nstreams; i++) {
	hipStreamCreate(&streams[i]);
        //printf("creating %d th stream\n", i);
    }
    for(int i = 0; i < nstreams; i++){
        printf("%d th stream is working \n", i);
        int start_per_stream = i * size_per_stream;
        printf("size_per_steram=%d, start_per_stream=%d\n",size_per_stream,start_per_stream);
        /* async copy data to GPU */
        hipMemcpyAsync(dx + start_per_stream, hx + start_per_stream, 
                    size_per_stream, hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(dy + start_per_stream, hy + start_per_stream, 
                    size_per_stream, hipMemcpyHostToDevice, streams[i]);

        /* launch GPU kernel */
        vec_add<<<grid, bs, 0, streams[i]>>>(dx + start_per_stream, dy + start_per_stream, dz + start_per_stream, size_per_stream); 
        
        /* async copy GPU result to CPU */
        hipMemcpyAsync(hz + start_per_stream, dz + start_per_stream, size_per_stream, hipMemcpyDeviceToHost, streams[i]);
    } 
    // when using ****Async, here we need use cudaDeviceSynchronize to sync host and all streams of device.
    // when only need sync one stream and device, use cudaStreamSynchronize, which is light-weight
    hipDeviceSynchronize();
    /* CPU compute */
    FLOAT* hz_cpu_res = (FLOAT *) malloc(nbytes);
    vec_add_cpu(hx, hy, hz_cpu_res, N);

    /* check GPU result with CPU*/
    for (int i = 0; i < N; ++i) {
        if (fabs(hz_cpu_res[i] - hz[i]) > 1e-6) {
            printf("index: %d, cpu: %f, gpu: %f\n", i, hz_cpu_res[i], hz[i]);
            break;
        }
    }
    printf("Result right\n");
    for (int i = 0; i < nstreams; i++) {
	hipStreamDestroy(streams[i]);
        printf("destroying %d th stream\n", i);
    }
    hipFree(dx);
    hipFree(dy);
    hipFree(dz);
    // free pinned memory
    hipHostFree(hx);
    hipHostFree(hy);
    hipHostFree(hz);
    free(hz_cpu_res);

    return 0;
}
