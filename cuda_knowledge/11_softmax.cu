#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "hip/hip_runtime.h"
#include <cmath>
#include <cassert>
#define WarpSize 32

bool CheckResult(float *out, float* groudtruth, int N){
    for (int i = 0; i < N; i++){
      if(i == 0){
        printf("1st comparsion: %f and %f \n" , out[i], groudtruth[i] );
      }
      if (out[i] != groudtruth[i]) {
          return false;
      }
    }
    return true;
}

float* softmaxCPU(float* input, float* result, int rows, int cols){
  for (int j = 0; j < rows; j++)
  {
    float total = 0;
    float MAX = 0;
    for(int i = 0; i < cols; i++)
    {
      MAX = max(input[j * cols + i], MAX);
    }
    for(int i = 0; i < cols; i++)
    {
      total += exp(input[j * cols + i] - MAX);
    }
    for(int i = 0; i < cols; i++)
    {
      result[j * cols + i] = exp(input[j * cols + i] - MAX) / total;
    }
  }

  return result;
}
template <typename T, int VecSize>
struct alignas(sizeof(T) * VecSize) VectorType {
  T val[VecSize];
};

template<typename T>
struct SumOp {
  __device__ __forceinline__ T operator()(const T& a, const T& b) const { return a + b; }
};

template<typename T>
struct MaxOp {
  __device__ __forceinline__ T operator()(const T& a, const T& b) const { return max(a, b); }
};

template<template<typename> class ReductionOp, typename T, int warp_width = WarpSize>
__inline__ __device__ T WarpReduce(T val) {
  for (int mask = warp_width / 2; mask > 0; mask /= 2) {
    // you can change L61 with __shfl_down_sync like 6_warp_level_reduce and see performance change
    val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask));
  }
  return val;
}

template<typename T>
__inline__ __device__ T Exp(T x);

template<>
__inline__ __device__ float Exp<float>(float x) {
  //return __expf(x);//fast math
  return exp(x);
}

template<typename T>
__inline__ __device__ T Inf();

template<>
__inline__ __device__ float Inf<float>() {
  return 10000000000;
}

template<typename T>
__inline__ __device__ T Div(T a, T b);

template<>
__inline__ __device__ float Div<float>(float a, float b) {
  //return __fdividef(a, b);//fast math
  return a / b;
}

template<int VecSize>
__device__ void load(const float* src, float* dst, int row, const int row_size, const int col) {
  using VecType = VectorType<float, VecSize>;
  const int offset = (row * row_size + col) / VecSize;
  *reinterpret_cast<VecType*>(dst) = *(reinterpret_cast<VecType*>(const_cast<float*>(src)) + offset);
}


template<int VecSize>
__device__ void store(float* dst, float* src, int row, const int row_size, const int col) {
  using VecType = VectorType<float, VecSize>;
  const int offset = (row * row_size + col) / VecSize;
  *(reinterpret_cast<VecType*>(dst) + offset) = *reinterpret_cast<VecType*>(src);
}


template<int pack_size, int cols_per_thread,
         int warp_width, int rows_per_thread>
__global__ void WarpSoftmax(const float* src, float* dst, const int rows, const int cols) {
  constexpr int num_packs = cols_per_thread / pack_size;
  assert(cols <= cols_per_thread * warp_width);
  float buf[rows_per_thread][cols_per_thread];
  const int global_warp_id = blockIdx.y * blockDim.y + threadIdx.y;
  const int num_global_warp = gridDim.y * blockDim.y;
  const int lane_id = threadIdx.x;
  const int step = num_global_warp * rows_per_thread;
  for (int row = global_warp_id * rows_per_thread; row < rows; row += step) { // 一个warp处理多行
    float thread_max[rows_per_thread];

    for (int row_id = 0; row_id < rows_per_thread; ++row_id) {
      thread_max[row_id] = -Inf<float>();
      float* row_buf = buf[row_id];

      for (int pack_id = 0; pack_id < num_packs; ++pack_id) {
        const int pack_offset = pack_id * pack_size;
        const int col = (pack_id * warp_width + lane_id) * pack_size;
        if (col < cols) {
          // load (row+row_id, col) data from src to reg row_buf
          load<pack_size>(src, row_buf + pack_offset, row + row_id, rows, col);

          for (int i = 0; i < pack_size; ++i) {
            thread_max[row_id] = max(thread_max[row_id], row_buf[pack_offset + i]);
          }
        } else {

          for (int i = 0; i < pack_size; ++i) { row_buf[pack_offset + i] = -Inf<float>(); }
        }
      }
    }
    float warp_max[rows_per_thread];

    for (int row_id = 0; row_id < rows_per_thread; ++row_id) {
      warp_max[row_id] = WarpReduce<MaxOp, float, warp_width>(thread_max[row_id]);
    }
    float thread_sum[rows_per_thread];

    for (int row_id = 0; row_id < rows_per_thread; ++row_id) {
      thread_sum[row_id] = 0;
      float* row_buf = buf[row_id];

      for (int i = 0; i < cols_per_thread; ++i) {
        row_buf[i] = Exp(row_buf[i] - warp_max[row_id]);
        thread_sum[row_id] += row_buf[i];
      }
    }
    float warp_sum[rows_per_thread];
    // reduce规约
    for (int row_id = 0; row_id < rows_per_thread; ++row_id) {
      warp_sum[row_id] = WarpReduce<SumOp, float, warp_width>(thread_sum[row_id]);
    }

    for (int row_id = 0; row_id < rows_per_thread; ++row_id) {
      float* row_buf = buf[row_id];
      // 计算div
      for (int i = 0; i < cols_per_thread; ++i) {
        row_buf[i] = Div(row_buf[i], warp_sum[row_id]);
      }
      // 计算完毕store
      for (int i = 0; i < num_packs; ++i) {
        const int col = (i * warp_width + lane_id) * pack_size;
        if (col < cols) {
          store<pack_size>(dst, row_buf + i * pack_size, row + row_id, rows, col);
        }
      }
    }
  }
}

int main(){
    float milliseconds = 0;
    const int N = 1000 * 1024;
    float *src = (float *)malloc(N * sizeof(float));
    float *d_src;
    hipMalloc((void **)&d_src, N * sizeof(float));

    //int gridSize = ;//2d block, blockx=32,blocky=num warps in a block,griddimy=block nums
    //int blockSize = 256;
    float *dst = (float*)malloc(N * sizeof(float));
    float *d_dst;
    hipMalloc((void **)&d_dst, N * sizeof(float));
    float *groudtruth = (float *)malloc(N * sizeof(float));

    for(int i = 0; i < N; i++){
        src[i] = 1;
    }

    groudtruth = softmaxCPU(src, dst, 1000, 1024);

    hipMemcpy(d_src, src, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(1, 125);//y轴125个block,
    dim3 Block(32, 8);//x轴32个threads组成一个warp访问一行,y轴8个threads,8*125=1000行,每个warp处理一行
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    WarpSoftmax<1, 1024 / 32, 32, 1><<<Grid, Block>>>(d_src, d_dst, 1000, 1024);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(dst, d_dst, N * sizeof(float), hipMemcpyDeviceToHost);
    bool is_right = CheckResult(dst, groudtruth, N);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i=0;i<10;i++){
            printf("%lf ",dst[i]);
        }
        printf("\n");
    }
    printf("WarpSoftmax latency = %f ms\n", milliseconds);

    hipFree(d_src);
    hipFree(d_dst);
    free(src);
    free(dst);
    free(groudtruth);
}
