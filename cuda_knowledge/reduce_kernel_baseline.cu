#include "bits/stdc++.h"
#include <hip/hip_runtime.h>

#include<iostream>

// latency = 1033.946167 ms
__global__ void reduce_baseline(const int* input, int* output, size_t n) {
    int sum = 0;
    for (size_t i = 0; i < n; ++i) {
      sum += input[i];
    }
    *output = sum;
  }

  bool CheckResult(int *out, int groudtruth, int n){
    //int res = 0;
    //for (int i = 0; i < n; i++){
    //    res += out[i];
    //}
    if (*out != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    //const int N = 32 * 1024 * 1024;
    const int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    //const int blockSize = 256;
    const int blockSize = 1;
    //int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);//used later
    int GridSize = 1;
    int *a = (int *)malloc(N * sizeof(int));
    int *d_a;
    hipMalloc((void **)&d_a, N * sizeof(int)); // 主机保存一个指向设备内存的指针

    int *out = (int*)malloc((GridSize) * sizeof(int));
    int *d_out;
    hipMalloc((void **)&d_out, (GridSize) * sizeof(int));

    // 初始化 a
    for (int i = 0; i < N; i++){
        a[i] = 1;
    }

    int groudtruth = N * 1;

    hipMemcpy( d_a, a, N * sizeof(int), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_baseline<<<1, 1>>>(d_a, d_out, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(int), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d\n", GridSize, N);

    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0; i < GridSize;i++){
            printf("res per block : %lf ",out[i]);
        }
        printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_baseline latency = %f ms\n", milliseconds);
    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}