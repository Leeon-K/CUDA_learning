#include <stdio.h>
#include <hip/hip_runtime.h>

#include <stdint.h>

#define ARRAY_SIZE 100000000   //Array size has to exceed L2 size to avoid L2 cache residence
#define MEMORY_OFFSET 10000000
#define BENCH_ITER 10  // 循环10次取平均
#define THREADS_NUM 256

// cuda支持向量化读写 两个用处：
// 1. 加速vectoradd 向量化load & store
// 2. 可以用来测量GPU显存带宽

__global__ void mem_bw(float *A, float *B, float *C){
    int tid = threadIdx.x + blockIdx.x * blockDim.x; //gridDim.x = 
    // printf("tid = %d, blockDim.x = %d,blockIdx.x=%d, gridDim.x = %d\n", tid, blockDim.x, blockIdx.x,gridDim.x);
    // for 线程数 小于 数据量                  多个blockDim.x * gridDim.x线程并行
    for (int i = tid; i < MEMORY_OFFSET / 4; i += blockDim.x * gridDim.x){
        float4 a1 = reinterpret_cast<float4*>(A)[i];
        float4 b1 = reinterpret_cast<float4*>(B)[i];
        float4 c1;
        // 测量显存带宽方法1:向量加法,3019GB/s
		c1.x = a1.x + b1.x;
		c1.y = a1.y + b1.y;
		c1.z = a1.z + b1.z;
		c1.w = a1.w + b1.w;
        reinterpret_cast<float4*>(C)[i] = c1;
    }
}


void vec_add_cpu(float *x, float *y, float *z, int N)
{
    for (int i = 0; i < 20; i++) z[i] = y[i] + x[i];
}


int main(){
	float *A = (float*) malloc(ARRAY_SIZE*sizeof(float));
	float *B = (float*) malloc(ARRAY_SIZE*sizeof(float));
	float *C = (float*) malloc(ARRAY_SIZE*sizeof(float));

	float *A_g;
	float *B_g;
	float *C_g;

	float milliseconds = 0;

	for (uint32_t i=0; i<ARRAY_SIZE; i++){
		A[i] = (float)i;
		B[i] = (float)i;
	}
	hipMalloc((void**)&A_g, ARRAY_SIZE*sizeof(float));
	hipMalloc((void**)&B_g, ARRAY_SIZE*sizeof(float));
	hipMalloc((void**)&C_g, ARRAY_SIZE*sizeof(float));

    hipMemcpy(A_g, A, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_g, B, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
  
	int BlockNums = MEMORY_OFFSET / 256; //gridDim.x
    //warm up to occupy L2 cache
	printf("warm up start\n");
    mem_bw<<<BlockNums, THREADS_NUM>>>(A_g, B_g, C_g); // 占满cache 后面就访问不到，直接对显存访问
    printf("warm up end\n");


	// time start using cudaEvent
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	for (int i = BENCH_ITER - 1; i >= 0; --i) {
		mem_bw<<<BlockNums / 4, THREADS_NUM>>>(A_g + i * MEMORY_OFFSET, B_g + i * MEMORY_OFFSET, C_g + i * MEMORY_OFFSET);
	}
	// time stop using cudaEvent
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);

	hipMemcpy(C, C_g, ARRAY_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	/* CPU compute */
	float* C_cpu_res = (float *) malloc(20*sizeof(float));
	vec_add_cpu(A, B, C_cpu_res, ARRAY_SIZE);

	/* check GPU result with CPU*/
	for (int i = 0; i < 20; ++i) {
		/* 测量显存带宽时, 修改C_cpu_res[i]为0 */
		if (fabs(C_cpu_res[i] - C[i]) > 1e-6) {
			printf("Result verification failed at element index %d!\n", i);
		}
	}
	printf("Result right\n");
	unsigned N = ARRAY_SIZE * 4;
	/* 测量显存带宽时, 根据实际读写的数组个数, 指定110行是1/2/3 */
	printf("Mem BW= %f (GB/sec)\n", 3 * (float)N / milliseconds / 1e6); //  835.263744 (GB/sec)
  	hipFree(A_g);
  	hipFree(B_g);
  	hipFree(C_g);

  	free(A);
  	free(B);
  	free(C);
  	free(C_cpu_res);
}
