#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <iostream>

#define M 8192
#define N 8192
#define K 8192
// 分块大小
#define BM 64
#define BN 64
#define BK 16
// #define A(i,j) A[(i) + (j)*lda]
// #define B(i,j) B[(i) + (j)*ldb]
// #define C(i,j) C[(i) + (j)*ldc]
#define IDX2C(i, j, ld) ((j) * (ld) + (i)) // columb-major
#define IDX2R(i, j, lr) ((i) * (lr) + (j)) // row-major
// #define vload(v1,addr) v1 = *((float4 *)(addr));
// #define vstore(addr,v1) *((float4 *)(addr)) = v1;
#define vload(v1,addr)\
    v1 = *((float4 *)(addr));
#define vstore(addr,v1)\
    *((float4 *)(addr)) = v1;
//v1 += v2 * s3, vector scaling
#define vscal(v1, v2, s3)\
    v1.x+=v2.x*s3;\
    v1.y+=v2.y*s3;\
    v1.z+=v2.z*s3;\
    v1.w+=v2.w*s3;
//v1 = alpha * v2 + beta * v3, simd fma
#define simd_axpby(v1, alpha, v2, beta, v3)\
    v1.x=alpha*v2.x+beta*v3.x;\
    v1.y=alpha*v2.y+beta*v3.y;\
    v1.z=alpha*v2.z+beta*v3.z;\
    v1.w=alpha*v2.w+beta*v3.w;

void cpuSgemm(const int m,const int n,const int k,const float* alpha, const float *A, const float *B,
    const float *beta, float* C){
    for (int idx_m = 0; idx_m < m; idx_m++){
        for (int idx_n = 0;idx_n < n; idx_n++){
            float sum = 0.0;
            for (int idx_k = 0; idx_k < k; idx_k++){
                sum += A[IDX2C(idx_m, idx_k, m)] * B[IDX2C(idx_k,idx_n,k)];
            }
            C[IDX2C(idx_m,idx_n,m)] = sum* *(alpha) + *(beta) *C[IDX2C(idx_m,idx_n,m)];
        }
    }
}
__global__ void naive_matmul(const int m,const int n,const int k,const float alpha, const float *A, const float *B, const float beta, float* C)
{
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;
    A = &A[IDX2C(bx<<5,0,m)]; // blockdim(32,32)
    B = &B[IDX2C(0,by<<5,k)];
    C = &C[IDX2C(bx<<5,by<<5,m)];
    float sum = 0.0;
    for (int i = 0; i < k; i++){
        sum += A[IDX2C(tx,i,m)] * B[IDX2C(i,ty,k)];
    }
    C[IDX2C(tx,ty,m)] = alpha * sum + beta * C[IDX2C(tx,ty,m)];
}

__global__ void gemm_submat4x4(const int m,const int n,const int k,const float alpha, const float *A, const float *B, const float beta, float* C)
{
    // 分配共享内存  
    __shared__ float sa[1024];
    __shared__ float sb[1024];

    int tx = threadIdx.x; // 一个block有256个线程
    int bx = blockIdx.x, by = blockIdx.y;
    // int row = tx&31, col = tx>>5; // 低位行x  高维列y
    int row_a = (tx & 15) << 2, col_a = tx >> 4; // 
    int row_b = (tx & 3) << 2, col_b = tx >> 2; //
    int row_c = (tx & 15) << 2,col_c = (tx >> 4) << 2;
    A = &A[IDX2C(bx<<6,0,m)]; //分为 64 64 的小块
    B = &B[IDX2C(0,by<<6,k)];
    C = &C[IDX2C(bx<<6,by<<6,m)];
    float4 Av, Bv, Cv[4], Cres[4];
    memset(Cres, 0, sizeof(Cres));
    for (int i = 0; i < k; i += BK){ // BK 变小 一个线程负责计算双倍
        // 存储是 列主序, vec load
        vload(Av, &A[IDX2C(row_a, col_a, m)]);
        vload(Bv, &B[IDX2C(row_b, col_b, k)]);
        ((float4 *)sa)[tx] = Av;  // sa转换为float4 一次读取四个
        sb[IDX2C(col_b,row_b,BN)] = Bv.x; // 一次读取四个
        sb[IDX2C(col_b,row_b+1,BN)] = Bv.y; 
        sb[IDX2C(col_b,row_b+2,BN)] = Bv.z; 
        sb[IDX2C(col_b,row_b+3,BN)] = Bv.w; 
        A += m << 4;
        B += 16; // 一次16行 
        __syncthreads();
        #pragma unroll
        for (int b_k = 0; b_k < BK; b_k++){
            vload(Av, &sa[IDX2C(row_c, b_k, BM)]) // 行分块
            vload(Bv, &sb[IDX2C(col_c, b_k, BN)]) // 列分块
            vscal(Cres[0], Av, Bv.x)
            vscal(Cres[1], Av, Bv.y)
            vscal(Cres[2], Av, Bv.z)
            vscal(Cres[3], Av, Bv.w)
        }
        __syncthreads();
    }
    vload(Cv[0], &C[IDX2C(row_c,col_c,m)])
    vload(Cv[1], &C[IDX2C(row_c,col_c+1,m)])
    vload(Cv[2], &C[IDX2C(row_c,col_c+2,m)])
    vload(Cv[3], &C[IDX2C(row_c,col_c+3,m)]) // 向量化读
    simd_axpby(Cres[0],alpha,Cres[0],beta,Cv[0])
    simd_axpby(Cres[1],alpha,Cres[1],beta,Cv[1])
    simd_axpby(Cres[2],alpha,Cres[2],beta,Cv[2])
    simd_axpby(Cres[3],alpha,Cres[3],beta,Cv[3])

    vstore(&C[IDX2C(row_c,col_c, m)], Cres[0])
    vstore(&C[IDX2C(row_c,col_c + 1, m)], Cres[1])
    vstore(&C[IDX2C(row_c,col_c + 2, m)], Cres[2])
    vstore(&C[IDX2C(row_c,col_c + 3, m)], Cres[3])  // 向量化写
}

void gpuSgemm(int m, int n, int k, const float *alpha, 
    const float *A, const float *B, const float *beta, float *C) {
        int blocksize = 256;
        // int GridSize = ceil(sqrt((N+bs-1.) / bs));
        // int GridSize = ceil((M*N+blocksize-1.) / blocksize);
        int gridx = floor(m/BM);
        int gridy = floor(n/BN);
        dim3 Grid(gridx, gridy); //
        dim3 Block(256); // 32 * 32 = 1024  
        //malloc on device
        float *devPtrA, *devPtrB, *devPtrC,*devPtrD;
        hipMalloc((void**)&devPtrA, sizeof(float) * m * k);
        hipMalloc((void**)&devPtrB, sizeof(float) * k * n);
        hipMalloc((void**)&devPtrC, sizeof(float) * m * n);
        hipMalloc((void**)&devPtrD, sizeof(float) * m * n);
        //copy A and B to device
        hipMemcpy(devPtrA, A, m * k * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(devPtrB, B, k * n * sizeof(float), hipMemcpyHostToDevice);
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
// ------------------------------------------------------------------------------------
        gemm_submat4x4<<<Grid,Block>>>(m,n,k,*alpha,devPtrA,devPtrB,*beta,devPtrC);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
    
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("gpu with gemm_shared_v2 kernel time:%f ms\n",milliseconds);
        float* matrix_out_cpu=(float*)malloc(sizeof(float) * M * N);
        float* matrix_out_gpu=(float*)malloc(sizeof(float) * M * N);
        hipMemcpy(matrix_out_cpu, devPtrC, m * n * sizeof(float), hipMemcpyDeviceToHost);
        dim3 Grid_n(m/32, n/32); //
        dim3 Block_n(32,32); // 32 * 32 = 1024  
        naive_matmul<<<Grid_n,Block_n>>>(m,n,k,*alpha,devPtrA,devPtrB,*beta,devPtrD);
        hipMemcpy(matrix_out_gpu, devPtrD, m * n * sizeof(float), hipMemcpyDeviceToHost);

        float EPSILON = 0.1;
        // check result                                             
        printf("check\n");
        for (int i = 0; i < M * N; ++i) {
            float error = (matrix_out_cpu[i] - matrix_out_gpu[i]) 
                / matrix_out_gpu[i];
            if (error < -EPSILON || error > EPSILON)
                printf("wrong, %f, %f, %f\n", matrix_out_cpu[i], matrix_out_gpu[i], 
                    error);
        }
        printf("right\n");

        //release memory on device
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        hipFree(devPtrD);
        free(matrix_out_cpu);
        free(matrix_out_gpu);
}

int main(){
    float rand_min = -10.0, rand_max = 10.0, rand_num = 0.0;

    float* matrix_in1 = (float*)malloc(sizeof(float) * M * K);
    float* matrix_in2 = (float*)malloc(sizeof(float) * K * N);
    float* matrix_out_cpu = (float*)malloc(sizeof(float) * M * N);
    float* matrix_out_gpu = (float*)malloc(sizeof(float) * M * N);

    for (int i = 0; i< M * K; i++){
        rand_num = (float)rand() / RAND_MAX; // RAND_MAX = 32767
        matrix_in1[i] = rand_min + rand_num * (rand_max - rand_min);
    }
    for (int i = 0; i < K * N; ++i) {
        rand_num = (float)rand()/RAND_MAX;
        matrix_in2[i] = rand_min + rand_num * (rand_max - rand_min);
    }

    clock_t start, stop;
    float a = 1.0, b = 0.0;
    double duration;
    
    // // record cpu execution time
    // start=clock();
    // cpuSgemm(M, N, K, &a, matrix_in1, matrix_in2, &b, matrix_out_cpu);
    // stop=clock();
    // duration=(double)(stop-start)/CLOCKS_PER_SEC;
    // printf("cpu time:%f\n",duration);

    ///////////////////////////////////////////////////////////////////////////////////
    gpuSgemm(M, N, K, &a, matrix_in1, matrix_in2, &b, matrix_out_gpu);
  
    // float EPSILON = 0.1;
    // // check result                                             
    // printf("check\n");
    // for (int i = 0; i < M * N; ++i) {
    //     float error = (matrix_out_cpu[i] - matrix_out_gpu[i]) 
    //         / matrix_out_gpu[i];
    //     if (error < -EPSILON || error > EPSILON)
    //         printf("wrong, %f, %f, %f\n", matrix_out_cpu[i], matrix_out_gpu[i], 
    //             error);
    // }
    // printf("right\n");

    //release memory on host
    free(matrix_in1);
    free(matrix_in2);
    free(matrix_out_cpu);
    free(matrix_out_gpu);

    return 0;
}