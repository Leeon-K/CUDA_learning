
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstddef>
#include <cstdint>
#include <iomanip>
#include <iostream>
#include <random>
#include <stdexcept>
#include <vector>

#define BLOCK_DIM 32

#define checkCuda(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

template <typename T>
std::vector<T> create_rand_vector(size_t n)
{
    std::random_device r;
    std::default_random_engine e(r());
    std::uniform_int_distribution<int> uniform_dist(-256, 256);

    std::vector<T> vec(n);
    for (size_t i{0}; i < n; ++i)
    {
        vec.at(i) = static_cast<T>(uniform_dist(e));
    }

    return vec;
}

// mat_1: m x n
// mat_2: n x p
// mat_3: m x p
template <typename T>
void mm(T const* mat_1, T const* mat_2, T* mat_3, size_t m, size_t n, size_t p)
{
    // Compute the cells in mat_3 sequentially.
    for (size_t i{0}; i < m; ++i)
    {
        for (size_t j{0}; j < p; ++j)
        {
            T acc_sum{0};
            for (size_t k{0}; k < n; ++k)
            {
                acc_sum += mat_1[i * n + k] * mat_2[k * p + j];
            }
            mat_3[i * p + j] = acc_sum;
        }
    }
}


template <typename T>
__global__ void mm_kernel(T const* mat_1, T const* mat_2, T* mat_3, size_t m,
                          size_t n, size_t p)
{
    // 2D block and 2D thread
    // Each thread computes one cell in mat_3.
    size_t row{blockIdx.y * blockDim.y + threadIdx.y};
    size_t col{blockIdx.x * blockDim.x + threadIdx.x};

    // Do not process outside the matrix.
    // Do not forget the equal sign!
    if ((row >= m) || (col >= p)) {
      return;
    }

    T acc_sum{0};
    for (size_t k{0}; k < n; ++k)
    {
      acc_sum += mat_1[row * n + k] * mat_2[k * p + col];
    }
    mat_3[row * p + col] = acc_sum;
}

#define TILE_SIZE 32
template <typename T>
__global__ void mm_tiled_kernel(T const *mat_1, T const *mat_2, T *mat_3,
                                size_t m, size_t n, size_t p) {
  __shared__ T tile_mat_1[TILE_SIZE][TILE_SIZE];
  __shared__ T tile_mat_2[TILE_SIZE][TILE_SIZE];

  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = by * blockDim.y + ty;
  int col = bx * blockDim.x + tx;

  T acc_sum{0};
  for (int k = 0; k < n / TILE_SIZE; k++) {
    tile_mat_1[ty][tx] = mat_1[row * n + k * TILE_SIZE + tx];
    tile_mat_2[ty][tx] = mat_2[(k * TILE_SIZE + ty) * p + col];
    __syncthreads();
    for (int i = 0; i < TILE_SIZE; ++i)
      acc_sum += tile_mat_1[ty][i] * tile_mat_2[i][tx];
    __syncthreads();
  }

  mat_3[row * p + col] = acc_sum;
}

template <typename T>
void mm_cuda(T const* mat_1, T const* mat_2, T* mat_3, size_t m, size_t n,
             size_t p)
{
    dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
    dim3 blocks_per_grid(1, 1);
    blocks_per_grid.x = std::ceil(static_cast<double>(p) /
                                  static_cast<double>(threads_per_block.x));
    blocks_per_grid.y = std::ceil(static_cast<double>(m) /
                                  static_cast<double>(threads_per_block.y));
    mm_kernel<<<blocks_per_grid, threads_per_block>>>(mat_1, mat_2, mat_3, m, n,
                                                      p);
}

template <typename T>
void mm_cuda_tiled(T const *mat_1, T const *mat_2, T *mat_3, size_t m, size_t n,
                   size_t p) {
  dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
  dim3 blocks_per_grid(1, 1);
  blocks_per_grid.x = std::ceil(static_cast<double>(p) /
                                static_cast<double>(threads_per_block.x));
  blocks_per_grid.y = std::ceil(static_cast<double>(m) /
                                static_cast<double>(threads_per_block.y));
  mm_tiled_kernel<<<blocks_per_grid, threads_per_block>>>(mat_1, mat_2, mat_3,
                                                          m, n, p);
}

template <typename T>
bool allclose(std::vector<T> const& vec_1, std::vector<T> const& vec_2,
              T const& abs_tol)
{
    if (vec_1.size() != vec_2.size())
    {
        return false;
    }
    for (size_t i{0}; i < vec_1.size(); ++i)
    {
        if (std::abs(vec_1.at(i) - vec_2.at(i)) > abs_tol)
        {
            std::cout << vec_1.at(i) << " " << vec_2.at(i) << std::endl;
            return false;
        }
    }
    return true;
}

template <typename T>
bool random_test_mm_cuda(size_t m, size_t n, size_t p)
{
    std::vector<T> const mat_1_vec{create_rand_vector<T>(m * n)};
    std::vector<T> const mat_2_vec{create_rand_vector<T>(n * p)};
    std::vector<T> mat_3_vec(m * p);
    std::vector<T> mat_4_vec(m * p);
    T const* mat_1{mat_1_vec.data()};
    T const* mat_2{mat_2_vec.data()};
    T* mat_3{mat_3_vec.data()};
    T* mat_4{mat_4_vec.data()};

    mm(mat_1, mat_2, mat_3, m, n, p);

    T *d_mat_1, *d_mat_2, *d_mat_4;

    // Allocate device buffer.
    checkCuda(hipMalloc(&d_mat_1, sizeof(T) * mat_1_vec.size()));
    checkCuda(hipMalloc(&d_mat_2, sizeof(T) * mat_2_vec.size()));
    checkCuda(hipMalloc(&d_mat_4, sizeof(T) * mat_4_vec.size()));

    // Copy data from host to device.
    checkCuda(hipMemcpy(d_mat_1, mat_1, sizeof(T) * mat_1_vec.size(),
                         hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_mat_2, mat_2, sizeof(T) * mat_2_vec.size(),
                         hipMemcpyHostToDevice));

    // Run matrix multiplication on GPU.
    mm_cuda_tiled(d_mat_1, d_mat_2, d_mat_4, m, n, p);
    hipDeviceSynchronize();
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Matrix Multiplication kernel failed to execute."
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }

    // Copy data from device to host.
    checkCuda(hipMemcpy(mat_4, d_mat_4, sizeof(T) * mat_4_vec.size(),
                         hipMemcpyDeviceToHost));

    // Free device buffer.
    checkCuda(hipFree(d_mat_1));
    checkCuda(hipFree(d_mat_2));
    checkCuda(hipFree(d_mat_4));

    return allclose<T>(mat_3_vec, mat_4_vec, 1e-4);
}


int main()
{
    size_t  m{1024}, n{1024}, p{1024};
    assert(random_test_mm_cuda<int32_t>(m,n,p));
    assert(random_test_mm_cuda<float>(m,n,p));
    assert(random_test_mm_cuda<double>(m,n,p));

}