#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <iostream>

#define M 2048
#define N 2048
#define K 2048
// 分块大小
#define BM 128
#define BN 128
#define BK 8
// #define A(i,j) A[(i) + (j)*lda]
// #define B(i,j) B[(i) + (j)*ldb]
#define C(i,j) C[(i) + (j)*ldc]
#define IDX2C(i, j, ld) ((j) * (ld) + (i)) // columb-major
#define IDX2R(i, j, lr) ((i) * (lr) + (j)) // row-major
// #define vload(v1,addr) v1 = *((float4 *)(addr));
// #define vstore(addr,v1) *((float4 *)(addr)) = v1;
#define vload(v1,addr)\
    v1 = *((float4 *)(addr));
#define vstore(addr,v1)\
    *((float4 *)(addr)) = v1;
//v1 += v2 * s3, vector scaling
#define vscal(v1, v2, s3)\
    v1.x+=v2.x*s3;\
    v1.y+=v2.y*s3;\
    v1.z+=v2.z*s3;\
    v1.w+=v2.w*s3;
//v1 = alpha * v2 + beta * v3, simd fma
#define simd_axpby(v1, alpha, v2, beta, v3)\
    v1.x=alpha*v2.x+beta*v3.x;\
    v1.y=alpha*v2.y+beta*v3.y;\
    v1.z=alpha*v2.z+beta*v3.z;\
    v1.w=alpha*v2.w+beta*v3.w;

void cpuSgemm(const int m,const int n,const int k,const float* alpha, const float *A, const float *B,
    const float *beta, float* C){
    for (int idx_m = 0; idx_m < m; idx_m++){
        for (int idx_n = 0;idx_n < n; idx_n++){
            float sum = 0.0;
            for (int idx_k = 0; idx_k < k; idx_k++){
                sum += A[IDX2C(idx_m, idx_k, m)] * B[IDX2C(idx_k,idx_n,k)];
            }
            C[IDX2C(idx_m,idx_n,m)] = sum* *(alpha) + *(beta) *C[IDX2C(idx_m,idx_n,m)];
        }
    }
}
__global__ void naive_matmul(const int m,const int n,const int k,const float alpha, const float *A, const float *B, const float beta, float* C)
{
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;
    A = &A[IDX2C(bx<<5,0,m)]; // blockdim(32,32)
    B = &B[IDX2C(0,by<<5,k)];
    C = &C[IDX2C(bx<<5,by<<5,m)];
    float sum = 0.0;
    for (int i = 0; i < k; i++){
        sum += A[IDX2C(tx,i,m)] * B[IDX2C(i,ty,k)];
    }
    C[IDX2C(tx,ty,m)] = alpha * sum + beta * C[IDX2C(tx,ty,m)];
}

__global__ void gemm_submat_warppara(const int m,const int n,const int k,const float alpha, const float *A, const float *B, const float beta, float* C)
{
    // 分配共享内存  
    __shared__ float sa[1024];
    __shared__ float sb[1024];
    int ldc = m;
    int tx = threadIdx.x; // 一个block有256个线程
    int bx = blockIdx.x, by = blockIdx.y;
    int warp_id = tx>>5; // 256 分为 8个warp
    int lane_id = tx&31;
    int warp_row = warp_id & 3, warp_col = warp_id >> 2; // warp级别的并行
    int row_w = lane_id&3, col_w = lane_id>>2;
    
    int row_c = (warp_row<<5) + (row_w<<3), col_c = (warp_col<<6) + (col_w<<3);
    int row_a = (tx&31)<<2, col_a = tx>>5;
    int row_b = (tx&1)<<2, col_b = tx>>1;
    
    A = &A[IDX2C(bx<<7,0,m)]; //分为 64 64 的小块
    B = &B[IDX2C(0,by<<7,k)];
    C = &C[IDX2C(bx<<7,by<<7,m)];
    float4 Av1, Av2, Bv1, Bv2,  Cv[16], Cres[16];
    memset(Cres, 0, sizeof(Cres));
    for (int i = 0; i < k; i += BK){
        // 存储是 列主序, vec load
        vload(Av1, &A[IDX2C(row_a, col_a, m)]);
        vload(Bv1, &B[IDX2C(row_b, col_b, k)]);
        ((float4 *)sa)[tx] = Av1;  // sa转换为float4 一次读取四个
        sb[IDX2C(col_b,row_b,BN)] = Bv1.x; // 一次读取四个
        sb[IDX2C(col_b,row_b+1,BN)] = Bv1.y; 
        sb[IDX2C(col_b,row_b+2,BN)] = Bv1.z; 
        sb[IDX2C(col_b,row_b+3,BN)] = Bv1.w; 
        A += m << 3;
        B += 8; // 一次8行 
        __syncthreads();
        // #pragma unroll
        for (int b_k = 0; b_k < BK; b_k++){
            // vload(Av, &sa[IDX2C(row_a, b_k, BM)]) // 行分块
            // vload(Bv, &sb[IDX2C(col_c, b_k, BN)]) // 列分块
            // vscal(Cres[0], Av, Bv.x)
            // vscal(Cres[1], Av, Bv.y)
            // vscal(Cres[2], Av, Bv.z)
            // vscal(Cres[3], Av, Bv.w)
            vload(Av1, &sa[IDX2C(row_c, b_k, BM)]) // 行分块
            vload(Av2, &sa[IDX2C(row_c+4, b_k, BM)]) // 列分块
            vload(Bv1, &sb[IDX2C(col_c, b_k, BN)]) // 行分块
            vload(Bv2, &sb[IDX2C(col_c+4, b_k, BN)]) // 列分块
            vscal(Cres[0], Av1, Bv1.x)
            vscal(Cres[1], Av2, Bv1.x)
            vscal(Cres[2], Av1, Bv1.y)
            vscal(Cres[3], Av2, Bv1.y)
            vscal(Cres[4], Av1, Bv1.z)
            vscal(Cres[5], Av2, Bv1.z)
            vscal(Cres[6], Av1, Bv1.w)
            vscal(Cres[7], Av2, Bv1.w)
            vscal(Cres[8], Av1, Bv2.x)
            vscal(Cres[9], Av2, Bv2.x)
            vscal(Cres[10], Av1, Bv2.y)
            vscal(Cres[11], Av2, Bv2.y)
            vscal(Cres[12], Av1, Bv2.z)
            vscal(Cres[13], Av2, Bv2.z)
            vscal(Cres[14], Av1, Bv2.w)
            vscal(Cres[15], Av2, Bv2.w)
        }
        __syncthreads();
    }
    vload(Cv[0], &C[IDX2C(row_c,col_c,m)])
    vload(Cv[1], &C[IDX2C(row_c+4,col_c,m)])
    vload(Cv[2], &C[IDX2C(row_c,col_c+1,m)])
    vload(Cv[3], &C[IDX2C(row_c+4,col_c+1,m)]) // 向量化读
    vload(Cv[4], &C[IDX2C(row_c,col_c+2,m)])
    vload(Cv[5], &C[IDX2C(row_c+4,col_c+2,m)])
    vload(Cv[6], &C[IDX2C(row_c,col_c+3,m)])
    vload(Cv[7], &C[IDX2C(row_c+4,col_c+3,m)]) // 向量化读
    vload(Cv[8], &C[IDX2C(row_c,col_c+4,m)])
    vload(Cv[9], &C[IDX2C(row_c+4,col_c+4,m)]) 
    vload(Cv[10], &C[IDX2C(row_c,col_c+5,m)])
    vload(Cv[11], &C[IDX2C(row_c+4,col_c+5,m)])
    vload(Cv[12], &C[IDX2C(row_c,col_c+6,m)])
    vload(Cv[13], &C[IDX2C(row_c+4,col_c+6,m)])
    vload(Cv[14], &C[IDX2C(row_c,col_c+7,m)])
    vload(Cv[15], &C[IDX2C(row_c+4,col_c+7,m)])
    
    for (int i = 0; i < 16; i++){
        simd_axpby(Cres[i],alpha,Cres[i],beta,Cv[i])
    }

    vstore(&C(row_c,col_c), Cres[0])
    vstore(&C(row_c+4,col_c), Cres[1])
    vstore(&C(row_c,col_c+1), Cres[2])
    vstore(&C(row_c+4,col_c+1), Cres[3])
    vstore(&C(row_c,col_c+2), Cres[4])
    vstore(&C(row_c+4,col_c+2), Cres[5])
    vstore(&C(row_c,col_c+3), Cres[6])
    vstore(&C(row_c+4,col_c+3), Cres[7])
    vstore(&C(row_c,col_c+4), Cres[8])
    vstore(&C(row_c+4,col_c+4), Cres[9])
    vstore(&C(row_c,col_c+5), Cres[10])
    vstore(&C(row_c+4,col_c+5), Cres[11])
    vstore(&C(row_c,col_c+6), Cres[12])
    vstore(&C(row_c+4,col_c+6), Cres[13])
    vstore(&C(row_c,col_c+7), Cres[14])
    vstore(&C(row_c+4,col_c+7), Cres[15])
}

void gpuSgemm(int m, int n, int k, const float *alpha, 
    const float *A, const float *B, const float *beta, float *C) {
        int blocksize = 256;
        // int GridSize = ceil(sqrt((N+bs-1.) / bs));
        // int GridSize = ceil((M*N+blocksize-1.) / blocksize);
        int gridx = floor(m/BM);
        int gridy = floor(n/BN);
        dim3 Grid(gridx, gridy); //
        dim3 Block(256); // 32 * 32 = 1024  
        //malloc on device
        float *devPtrA, *devPtrB, *devPtrC,*devPtrD;
        hipMalloc((void**)&devPtrA, sizeof(float) * m * k);
        hipMalloc((void**)&devPtrB, sizeof(float) * k * n);
        hipMalloc((void**)&devPtrC, sizeof(float) * m * n);
        hipMalloc((void**)&devPtrD, sizeof(float) * m * n);
        //copy A and B to device
        hipMemcpy(devPtrA, A, m * k * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(devPtrB, B, k * n * sizeof(float), hipMemcpyHostToDevice);
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
// ------------------------------------------------------------------------------------
        gemm_submat_warppara<<<Grid,Block>>>(m,n,k,*alpha,devPtrA,devPtrB,*beta,devPtrC);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
    
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("gpu with gemm_shared_v2 kernel time:%f ms\n",milliseconds);
        float* matrix_out_cpu=(float*)malloc(sizeof(float) * M * N);
        float* matrix_out_gpu=(float*)malloc(sizeof(float) * M * N);
        hipMemcpy(matrix_out_cpu, devPtrC, m * n * sizeof(float), hipMemcpyDeviceToHost);
        dim3 Grid_n(m/32, n/32); //
        dim3 Block_n(32,32); // 32 * 32 = 1024  
        naive_matmul<<<Grid_n,Block_n>>>(m,n,k,*alpha,devPtrA,devPtrB,*beta,devPtrD);
        hipMemcpy(matrix_out_gpu, devPtrD, m * n * sizeof(float), hipMemcpyDeviceToHost);

        float EPSILON = 0.1;
        // check result                                             
        printf("check\n");
        for (int i = 0; i < M * N; ++i) {
            float error = (matrix_out_cpu[i] - matrix_out_gpu[i]) 
                / matrix_out_gpu[i];
            if (error < -EPSILON || error > EPSILON)
                printf("wrong, %f, %f, %f\n", matrix_out_cpu[i], matrix_out_gpu[i], 
                    error);
        }
        printf("right\n");

        //release memory on device
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        hipFree(devPtrD);
        free(matrix_out_cpu);
        free(matrix_out_gpu);
}

int main(){
    float rand_min = -10.0, rand_max = 10.0, rand_num = 0.0;

    float* matrix_in1 = (float*)malloc(sizeof(float) * M * K);
    float* matrix_in2 = (float*)malloc(sizeof(float) * K * N);
    float* matrix_out_cpu = (float*)malloc(sizeof(float) * M * N);
    float* matrix_out_gpu = (float*)malloc(sizeof(float) * M * N);

    for (int i = 0; i< M * K; i++){
        rand_num = (float)rand() / RAND_MAX; // RAND_MAX = 32767
        matrix_in1[i] = rand_min + rand_num * (rand_max - rand_min);
    }
    for (int i = 0; i < K * N; ++i) {
        rand_num = (float)rand()/RAND_MAX;
        matrix_in2[i] = rand_min + rand_num * (rand_max - rand_min);
    }

    clock_t start, stop;
    float a = 1.0, b = 0.0;
    double duration;
    
    // // record cpu execution time
    // start=clock();
    // cpuSgemm(M, N, K, &a, matrix_in1, matrix_in2, &b, matrix_out_cpu);
    // stop=clock();
    // duration=(double)(stop-start)/CLOCKS_PER_SEC;
    // printf("cpu time:%f\n",duration);

    ///////////////////////////////////////////////////////////////////////////////////
    gpuSgemm(M, N, K, &a, matrix_in1, matrix_in2, &b, matrix_out_gpu);
  
    // float EPSILON = 0.1;
    // // check result                                             
    // printf("check\n");
    // for (int i = 0; i < M * N; ++i) {
    //     float error = (matrix_out_cpu[i] - matrix_out_gpu[i]) 
    //         / matrix_out_gpu[i];
    //     if (error < -EPSILON || error > EPSILON)
    //         printf("wrong, %f, %f, %f\n", matrix_out_cpu[i], matrix_out_gpu[i], 
    //             error);
    // }
    // printf("right\n");

    //release memory on host
    free(matrix_in1);
    free(matrix_in2);
    free(matrix_out_cpu);
    free(matrix_out_gpu);

    return 0;
}