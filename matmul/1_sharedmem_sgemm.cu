#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <iostream>

#define M 1024
#define N 1024
#define K 1024
// 分块大小
#define BM 32
#define BN 32
#define BK 32
// #define A(i,j) A[(i) + (j)*lda]
// #define B(i,j) B[(i) + (j)*ldb]
// #define C(i,j) C[(i) + (j)*ldc]
#define IDX2C(i, j, ld) ((j) * (ld) + (i)) // columb-major
#define IDX2R(i, j, ld) ((i) * (ld) + (j)) // row-major
void cpuSgemm(const int m,const int n,const int k,const float* alpha, const float *A, const float *B,
    const float *beta, float* C){
    for (int idx_m = 0; idx_m < m; idx_m++){
        for (int idx_n = 0;idx_n < n; idx_n++){
            float sum = 0.0;
            for (int idx_k = 0; idx_k < k; idx_k++){
                sum += A[IDX2C(idx_m, idx_k, m)] * B[IDX2C(idx_k,idx_n,k)];
            }
            C[IDX2C(idx_m,idx_n,m)] = sum* *(alpha) + *(beta) *C[IDX2C(idx_m,idx_n,m)];
        }
    }
}
__global__ void naive_matmul(const int m,const int n,const int k,const float alpha, const float *A, const float *B, const float beta, float* C)
{
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;
    A = &A[IDX2C(bx<<5,0,m)]; // blockdim(32,32)
    B = &B[IDX2C(0,by<<5,k)];
    C = &C[IDX2C(bx<<5,by<<5,m)];
    float sum = 0.0;
    for (int i = 0; i < k; i++){
        sum += A[IDX2C(tx,i,m)] * B[IDX2C(i,ty,k)];
    }
    C[IDX2C(tx,ty,m)] = alpha * sum + beta * C[IDX2C(tx,ty,m)];
}

__global__ void gemm_shared_v2(const int m,const int n,const int k,const float alpha, const float *A, const float *B, const float beta, float* C)
{
    // 分配共享内存  
    __shared__ float sa[BM*BK];
    __shared__ float sb[BK*BN];

    int tx = threadIdx.x, ty = threadIdx.y; // 一个block有32*32个线程
    int bx = blockIdx.x, by = blockIdx.y;
    A = &A[IDX2C(bx<<5,0,m)]; // blockdim(M/32,N/32)
    B = &B[IDX2C(0,by<<5,k)]; // 分块
    C = &C[IDX2C(bx<<5,by<<5,m)];
    float sum = 0.0;
    for (int i = 0; i < k; i += BK){
        sa[IDX2R(tx,ty,BM)] = A[IDX2C(tx,ty,m)];
        sb[IDX2R(ty,tx,BK)] = B[IDX2C(tx,ty,k)]; // 一次读取一个方块
        A += m<<5; // 一行 一次32行
        B += 32; //小方块一列32
        __syncthreads();
        for (int b_k = 0; b_k < BK; b_k++){
            sum += sa[IDX2R(tx,b_k,BM)] * sb[IDX2R(ty,b_k,BK)];
        }
        __syncthreads();
    }
    C[IDX2C(tx,ty,m)] = alpha * sum + beta * C[IDX2C(tx,ty,m)];
}

void gpuSgemm(int m, int n, int k, const float *alpha, 
    const float *A, const float *B, const float *beta, float *C) {
        int blocksize = 256;
        // int GridSize = ceil(sqrt((N+bs-1.) / bs));
        // int GridSize = ceil((M*N+blocksize-1.) / blocksize);
        int gridx = floor(M/32);
        int gridy = floor(N/32);
        dim3 Grid(gridx, gridy); //
        dim3 Block(32, 32); // 32 * 32 = 1024  
        //malloc on device
        float *devPtrA, *devPtrB, *devPtrC,*devPtrD;
        hipMalloc((void**)&devPtrA, sizeof(float) * m * k);
        hipMalloc((void**)&devPtrB, sizeof(float) * k * n);
        hipMalloc((void**)&devPtrC, sizeof(float) * m * n);
        hipMalloc((void**)&devPtrD, sizeof(float) * m * n);
        //copy A and B to device
        hipMemcpy(devPtrA, A, m * k * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(devPtrB, B, k * n * sizeof(float), hipMemcpyHostToDevice);
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
// ------------------------------------------------------------------------------------
        gemm_shared_v2<<<Grid,Block>>>(m,n,k,*alpha,devPtrA,devPtrB,*beta,devPtrC);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
    
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("gpu with gemm_shared_v2 kernel time:%f ms\n",milliseconds);
        float* matrix_out_cpu=(float*)malloc(sizeof(float) * M * N);
        float* matrix_out_gpu=(float*)malloc(sizeof(float) * M * N);
        hipMemcpy(matrix_out_cpu, devPtrC, m * n * sizeof(float), hipMemcpyDeviceToHost);
        naive_matmul<<<Grid,Block>>>(m,n,k,*alpha,devPtrA,devPtrB,*beta,devPtrD);
        hipMemcpy(matrix_out_gpu, devPtrD, m * n * sizeof(float), hipMemcpyDeviceToHost);

        float EPSILON = 0.1;
        // check result                                             
        printf("check\n");
        for (int i = 0; i < M * N; ++i) {
            float error = (matrix_out_cpu[i] - matrix_out_gpu[i]) 
                / matrix_out_gpu[i];
            if (error < -EPSILON || error > EPSILON)
                printf("wrong, %f, %f, %f\n", matrix_out_cpu[i], matrix_out_gpu[i], 
                    error);
        }
        printf("right\n");

        //release memory on device
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        hipFree(devPtrD);
        free(matrix_out_cpu);
        free(matrix_out_gpu);
}

int main(){
    float rand_min = -10.0, rand_max = 10.0, rand_num = 0.0;

    float* matrix_in1 = (float*)malloc(sizeof(float) * M * K);
    float* matrix_in2 = (float*)malloc(sizeof(float) * K * N);
    float* matrix_out_cpu = (float*)malloc(sizeof(float) * M * N);
    float* matrix_out_gpu = (float*)malloc(sizeof(float) * M * N);

    for (int i = 0; i< M * K; i++){
        rand_num = (float)rand() / RAND_MAX; // RAND_MAX = 32767
        matrix_in1[i] = rand_min + rand_num * (rand_max - rand_min);
    }
    for (int i = 0; i < K * N; ++i) {
        rand_num = (float)rand()/RAND_MAX;
        matrix_in2[i] = rand_min + rand_num * (rand_max - rand_min);
    }

    clock_t start, stop;
    float a = 1.0, b = 0.0;
    double duration;
    
    // // record cpu execution time
    // start=clock();
    // cpuSgemm(M, N, K, &a, matrix_in1, matrix_in2, &b, matrix_out_cpu);
    // stop=clock();
    // duration=(double)(stop-start)/CLOCKS_PER_SEC;
    // printf("cpu time:%f\n",duration);

    ///////////////////////////////////////////////////////////////////////////////////
    gpuSgemm(M, N, K, &a, matrix_in1, matrix_in2, &b, matrix_out_gpu);
  
    // float EPSILON = 0.1;
    // // check result                                             
    // printf("check\n");
    // for (int i = 0; i < M * N; ++i) {
    //     float error = (matrix_out_cpu[i] - matrix_out_gpu[i]) 
    //         / matrix_out_gpu[i];
    //     if (error < -EPSILON || error > EPSILON)
    //         printf("wrong, %f, %f, %f\n", matrix_out_cpu[i], matrix_out_gpu[i], 
    //             error);
    // }
    // printf("right\n");

    //release memory on host
    free(matrix_in1);
    free(matrix_in2);
    free(matrix_out_cpu);
    free(matrix_out_gpu);

    return 0;
}