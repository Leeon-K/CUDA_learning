#include "hip/hip_runtime.h"
__global__ void mysgemm_v7(int m, int n, int k, float alpha, float* A, float* B, float beta, float* C)
{
    int lda = M, ldb = K, ldc = M;
    int tx = threadIdx.x;
    int bx = blockIdx.x, by = blockIdx.y;
    int row_a = (tx&15)<<2, col_a = tx>>4;
    int row_b = (tx&3)<<2, col_b = tx>>2;
    int col_c = col_a<<2;
    int lda16 = lda<<4;
    A = &A((bx<<6),0);
    B = &B(0,(by<<6));
    C = &C((bx<<6),(by<<6));//the TB size is 64.
    __shared__ float sa7[1024];
    __shared__ float sb7[1024];
    float4 Av, Bv, Cv[4], Cres[4];
    memset(Cres, 0, sizeof(Cres)); //
    for (int k_count = 0; k_count<K; k_count+=KS_7){
        vload(Av, &A(row_a,col_a))
        vload(Bv, &B(row_b,col_b))
        ((float4 *)sa7)[tx] = Av;
        sb7(col_b,row_b)=Bv.x;
        sb7(col_b,row_b+1)=Bv.y;
        sb7(col_b,row_b+2)=Bv.z;
        sb7(col_b,row_b+3)=Bv.w;
        A+=lda16;B+=16;
        __syncthreads();
        #pragma unroll
        for (int inner_k_count=0;inner_k_count<KS_7;inner_k_count++){
            vload(Av, &sa7(row_a,inner_k_count))
            vload(Bv, &sb7(col_c,inner_k_count))
            vscal(Cres[0], Av, Bv.x)
            vscal(Cres[1], Av, Bv.y)
            vscal(Cres[2], Av, Bv.z)
            vscal(Cres[3], Av, Bv.w)
        }
        __syncthreads();
    }
    vload(Cv[0], &C(row_a,col_c))
    vload(Cv[1], &C(row_a,col_c+1))
    vload(Cv[2], &C(row_a,col_c+2))
    vload(Cv[3], &C(row_a,col_c+3))
    simd_axpby(Cres[0],alpha,Cres[0],beta,Cv[0])
    simd_axpby(Cres[1],alpha,Cres[1],beta,Cv[1])
    simd_axpby(Cres[2],alpha,Cres[2],beta,Cv[2])
    simd_axpby(Cres[3],alpha,Cres[3],beta,Cv[3])

    vstore(&C(row_a,col_c), Cres[0])
    vstore(&C(row_a,col_c+1), Cres[1])
    vstore(&C(row_a,col_c+2), Cres[2])
    vstore(&C(row_a,col_c+3), Cres[3])
}