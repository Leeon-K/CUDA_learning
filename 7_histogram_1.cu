#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

//5.158ms
template <int blockSize>
__global__ void histgram(int *hist_data, int *bin_data, int N)
{
    __shared__ int cache[256];
    int gtid = blockIdx.x * blockSize + threadIdx.x;
    int tid = threadIdx.x;
    cache[tid] = 0;//每个thread初始化shared mem
    __syncthreads();
    for (int i = gtid; i < N; i += gridDim.x * blockSize)
    {
        int val = hist_data[i];//每个单线程计算全局内存中的若干个值
        atomicAdd(&cache[val], 1); // bank conflict
    }
    __syncthreads();//此刻每个block的bin都已统计在cache这个smem中
    atomicAdd(&bin_data[tid], cache[tid]);
}

bool CheckResult(int *out, int* groudtruth, int N){
    for (int i = 0; i < N; i++){
        if (out[i] != groudtruth[i]) {
            return false;
        }
    }
    return true;
}

int main(){
    float milliseconds = 0;
    const int N = 25600000;
    int *hist = (int *)malloc(N * sizeof(int));
    int *bin = (int *)malloc(256 * sizeof(int));
    int *bin_data;
    int *hist_data;
    hipMalloc((void **)&bin_data, 256 * sizeof(int));
    hipMalloc((void **)&hist_data, N * sizeof(int));

    for(int i = 0; i < N; i++){
        hist[i] = i % 256;
    }

    int *groudtruth = (int *)malloc(256 * sizeof(int));;
    for(int j = 0; j < 256; j++){
        groudtruth[j] = 100000;
    }

    hipMemcpy(hist_data, hist, N * sizeof(int), hipMemcpyHostToDevice);
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    dim3 Grid(GridSize);
    dim3 Block(blockSize);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    histgram<blockSize><<<Grid, Block>>>(hist_data, bin_data, 256);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(bin, bin_data, 256 * sizeof(int), hipMemcpyDeviceToHost);
    bool is_right = CheckResult(bin, groudtruth, N);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0; i < 256; i++){
            printf("%lf ", bin[i]);
        }
        printf("\n");
    }
    printf("histogram + shared_mem + multi_value latency = %f ms\n", milliseconds);    

    hipFree(bin_data);
    hipFree(hist_data);
    free(bin);
    free(hist);
}
