#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


// latency = 0.121856 ms
__device__ void WarpSharedMemReduce(volatile float* smem, int tid){
    float x = smem[tid]; // 读取全局内存到shared mem，每个线程有独立的调用栈, 不同步
    if (blockDim.x >= 64) {
      x += smem[tid + 32]; __syncwarp();
      smem[tid] = x; __syncwarp();
    }
    x += smem[tid + 16]; __syncwarp();
    smem[tid] = x; __syncwarp();
    x += smem[tid + 8]; __syncwarp();
    smem[tid] = x; __syncwarp();
    x += smem[tid + 4]; __syncwarp();
    smem[tid] = x; __syncwarp();
    x += smem[tid + 2]; __syncwarp();
    smem[tid] = x; __syncwarp();
    x += smem[tid + 1]; __syncwarp();
    smem[tid] = x; __syncwarp();
}
// Note: using blockSize as a template arg can benefit from NVCC compiler optimization, 
// which is better than using blockDim.x that is known in runtime.
template<int blockSize>
__global__ void reduce_v4(float *d_in,float *d_out){
    __shared__ float smem[blockSize];

    unsigned int tid = threadIdx.x;
    // 仅改动这里  每个线程加载两个元素到shared mem对应位置
    unsigned int gtid = blockIdx.x * (blockSize*2) + threadIdx.x;
    // load: 每个线程加载两个元素到shared mem对应位置
    smem[tid] = d_in[gtid] + d_in[gtid + blockSize];
    __syncthreads();

    // compute: reduce in shared mem
    // 思考这里是如何并行的
    for (unsigned int index = blockDim.x / 2; index > 32; index >>= 1) {
        if (tid < index) {
            smem[tid] += smem[tid + index];
        }
        __syncthreads();
    }

    // __syncthreads 耗时多，最后一个warp 不需要
    // last warp 32拎出来单独作reduce
    if (tid < 32) {
        WarpSharedMemReduce(smem, tid);
    }
    // store: write back to global mem
    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}

bool CheckResult(float *out, float groudtruth, int n){
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    //const int N = 32 * 1024 * 1024;
    const int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    //int GridSize = 100000;
    float *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float *out = (float*)malloc((GridSize) * sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out, (GridSize) * sizeof(float));

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
    }

    float groudtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize / 2);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v4<blockSize / 2><<<Grid,Block>>>(d_a, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        //for(int i = 0; i < GridSize;i++){
            //printf("res per block : %lf ",out[i]);
        //}
        //printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_v3 latency = %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}
