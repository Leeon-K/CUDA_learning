#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <hip/hip_fp16.h>
#include "hip/hip_runtime.h"
typedef __half half;
typedef __half2 half2;

template<typename T>
struct MaskScaleAndElementwiseAddFunctor {
  MaskScaleAndElementwiseAddFunctor(const uint8_t* mask, const T* add_val, float scale)
      : mask(mask), add_val(add_val), scale(scale) {}
  __device__ T Compute(T x, int64_t i) const {
    return x * static_cast<T>(static_cast<bool>(mask[i]) * scale) + add_val[i];
  }
  const uint8_t* mask;
  const T* add_val;
  float scale;
};

template<>
struct MaskScaleAndElementwiseAddFunctor<half> {
  MaskScaleAndElementwiseAddFunctor(const uint8_t* mask, const half* add_val, float scale)
      : mask(mask), add_val(add_val), scale(scale) {}
  __device__ half Compute(half x, int64_t i) const {
    return x * static_cast<half>(static_cast<bool>(mask[i]) * scale) + add_val[i];
  }
  __device__ half2 ComputeHalf2(half2 x, int64_t i) const {
    const char2* mask_c2 = reinterpret_cast<const char2*>(mask);
    const half2* add_val_h2 = reinterpret_cast<const half2*>(add_val);
    char2 mask_val = mask_c2[i];
    half2 one_or_zero_h2;
    half2 h2_scale = __float2half2_rn(scale);
    one_or_zero_h2.x = mask_val.x;
    one_or_zero_h2.y = mask_val.y;
    return __hadd2(__hmul2(__hmul2(x, one_or_zero_h2), h2_scale), add_val_h2[i]);
  }
  const uint8_t* mask;
  const half* add_val;
  float scale;
};


template<typename FUNCTOR>
__global__ void FusedBiasAddCUDAKernelHalf2(FUNCTOR functor, const int elem_cnt,
                                        const int bias_size, const half* x, const half* bias,
                                        half* y) {
  const int h2_elem_cnt = elem_cnt / 2;
  const int h2_bias_size = bias_size / 2;
  const auto* x_h2 = reinterpret_cast<const half2*>(x);
  const auto* bias_h2 = reinterpret_cast<const half2*>(bias);
  auto* y_h2 = reinterpret_cast<half2*>(y);
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < h2_elem_cnt;
       i += blockDim.x * gridDim.x){
    half2 x_i = __hadd2(x_h2[i], bias_h2[i % h2_bias_size]);
    y_h2[i] = functor.ComputeHalf2(x_i, i);
  }
}

int main(){
    int ele_cnt = 1000;
    float scale = 0.5;
    uint8_t* mask_tensor = new uint8_t[1000];
    float* add_val = new float[1000];
    for (int i = 0; i < 1000; i++){
        mask_tensor[i] = (uint8_t)(i);
        add_val[i] = (float)(i);
    }
    int bias_size = 10;
 
    __half *x = (__half*) malloc(sizeof(__half) * ele_cnt);
    __half *y = (__half*) malloc(sizeof(__half) * ele_cnt);
    __half *bias = (__half*) malloc(sizeof(__half) * bias_size);
    for (int i = 0; i < ele_cnt; i++)
    {
      x[i] = (__half)(i);
    }
    __half *d_x, *d_y, *bias;
    hipMalloc((void **)&d_x, ele_cnt * sizeof(__half));
    hipMalloc((void **)&d_y, ele_cnt * sizeof(__half));
    hipMalloc((void **)&d_bias, bias_size * sizeof(__half));
    hipMemcpy(d_x, x, sizeof(__half) * ele_cnt, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(__half) * ele_cnt, hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, sizeof(__half) * bias_size, hipMemcpyHostToDevice);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int maxblocks = deviceProp.maxGridSize[0];
    int blockSize = 256;
    int gridSize = std::min((ele_cnt + blockSize - 1) / blockSize, maxblocks);
    MaskScaleAndElementwiseAddFunctor<half> mask_scale_elementwise_add_func(mask_tensor, add_val, scale);
    FusedBiasAddCUDAKernelHalf2<<<gridSize ,blockSize>>>(mask_scale_elementwise_add_func, ele_cnt, bias_size, d_x, d_bias, d_y);
    hipMemcpy(y, d_y, sizeof(__half) * ele_cnt, hipMemcpyDeviceToHost);
    
    free(x);
    free(y);
    free(bias);
    delete add_val;
    add_val = nullptr;
    delete mask_tensor;
    mask_tensor = nullptr;
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_bias);
}
