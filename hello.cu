#include <stdio.h>
#include <hip/hip_runtime.h>

#include<iostream>

// __global__  是CUDA kernel函数前缀，该函数被CPU调用启动，在GPU上执行
// blockidx.x 是 block的ID；  blockDim.x 是 block内线程数量； threadIdx.x 是线程的id 
__global__ void hello_cuda(){
    // threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("[%d] hello cuda from {blockIdx %d,blockDim %d,threadIdx %d,griddim.x=%d}\n", 
    idx, blockIdx.x, blockDim.x, threadIdx.x, gridDim.x);
}

int main(){
    //<<<>>> 启动CUDA kernel的标志 第一个数表示分配的block数量，第二个表示每个block中的线程数量
    // myKernel<<<gridSize, blockDim>>>  gridSize  blockDim 都可以是多维的
    hello_cuda<<<3,4>>>();  //     griddim.x  .y                       blockdim
    // 该函数处强制CPU等待GPU上的CUDA kernel执行，即同步，这里也可以不用写，只不过CPU会比GPU先执行完罢了
    // cudaDeviceSynchronize();
    return 0;
}