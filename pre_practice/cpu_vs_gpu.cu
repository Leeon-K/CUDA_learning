#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#ifdef WIN32
#include <windows.h>
#else
#include <sys/time.h>
#endif
#ifdef WIN32
int gettimeofday(struct timeval *tp, void *tzp)
{
  time_t clock;
  struct tm tm;
  SYSTEMTIME wtm;
  GetLocalTime(&wtm);
  tm.tm_year   = wtm.wYear - 1900;
  tm.tm_mon   = wtm.wMonth - 1;
  tm.tm_mday   = wtm.wDay;
  tm.tm_hour   = wtm.wHour;
  tm.tm_min   = wtm.wMinute;
  tm.tm_sec   = wtm.wSecond;
  tm. tm_isdst  = -1;
  clock = mktime(&tm);
  tp->tv_sec = clock;
  tp->tv_usec = wtm.wMilliseconds * 1000;
  return (0);
}
#endif
using namespace std;
// CPU version
int cpu_v()
{
    struct timeval start, end;
    gettimeofday( &start, NULL );
    float*A, *B, *C;
    int n = 1024 * 1024;
    int size = n * sizeof(float);
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    for(int i=0;i<n;i++)
    {
        A[i] = 90.0;
        B[i] = 10.0;
    }
    
    for(int i=0;i<n;i++)
    {
        C[i] = A[i] + B[i];
    }

    float max_error = 0.0;
    for(int i=0;i<n;i++)
    {
        max_error += fabs(100.0-C[i]);
    }
    cout << "max_error is " << max_error << endl;
    gettimeofday( &end, NULL );
    int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    cout << "total time is " << timeuse/1000 << "ms" <<endl;
    return 0;
}

// GPU version block 中的 thread 个数为 1024 充分利用SM
__global__ void Plus(float A[], float B[], float C[], int n)  // host 端调用 kernel 函数时，需要指定 grid 和 block 的大小
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    C[i] = A[i] + B[i];
}

int gpu_v()
{
    struct timeval start, end;
    gettimeofday( &start, NULL );
    float*A, *Ad, *B, *Bd, *C, *Cd;
    int n = 1024 * 1024;
    int size = n * sizeof(float);

    // CPU端分配内存
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    // 初始化数组
    for(int i=0;i<n;i++)
    {
        A[i] = 90.0;
        B[i] = 10.0;
    }
    // GPU端分配内存
    hipMalloc((void**)&Ad, size);
    hipMalloc((void**)&Bd, size);
    hipMalloc((void**)&Cd, size);

    
    // CPU的数据拷贝到GPU端
    hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
    hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);
    hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);
    // 定义kernel执行配置，（1024*1024/1024）个block，每个block里面有1024个线程
    // 线程多反而慢 因为
    dim3 dimBlock(512);
    dim3 dimGrid(n/512);

    // 执行kernel
    Plus<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, n);

    // 将在GPU端计算好的结果拷贝回CPU端
    hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);

    // 校验误差
    float max_error = 0.0;
    for(int i=0;i<n;i++)
    {
        max_error += fabs(100.0 - C[i]);
    }

    cout << "max error is " << max_error << endl;

    // 释放CPU端、GPU端的内存
    free(A);
    free(B);
    free(C);
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);
    gettimeofday( &end, NULL );
    int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    cout << "total time is " << timeuse/1000 << "ms" <<endl;
    return 0;
}


int main()
{
    // cpu_v();
    gpu_v();
}