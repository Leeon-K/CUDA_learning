#include "bits/stdc++.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
using namespace std;

//latency =  0.542720 ms
template<int blockSize>
__global__ void reduce_v0(float *d_in,float *d_out){
    // for (int i=0;i<blockSize;i++){
    //     if (i<2){printf("d_in[%d]: %f\n", i, d_in[i]);
    // }
    __shared__ float smem[blockSize]; // blocksize = 256

    int tid = threadIdx.x;
    // int gtid = blockIdx.x * blockSize + threadIdx.x;
    // load: 每个线程加载一个元素到shared mem对应位置
    smem[tid] = d_in[tid];
    __syncthreads(); // 把sharememory同步一下

    // compute: reduce in shared mem
    // 引入了 shared memory
    for(int index = 1; index < blockDim.x; index *= 2) {
        if (tid % (2 * index) == 0) { // 第一轮只有0 2 4 6 8 10 12 14 16 18 20 22 24 26 28 30这些线程会执行 // 后面0 4 8 16 32 64 128 256
            smem[tid] += smem[tid + index];
            // printf("tid: %d,index %d, smem[tid]: %f, smem[tid + index]: %f\n", tid,index, smem[tid], smem[tid + index]);
        }
        __syncthreads();
    }

    // store: write back to global mem
    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}

bool CheckResult(float *out, float groudtruth, int n){
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    const int N = 25600000;
    // const int N = 257;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    // const int blockSize = 4;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    //int GridSize = 100000;
    float *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float *out = (float*)malloc((GridSize) * sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out, (GridSize) * sizeof(float));

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
    }
    // a[N-1] = 2.0f
    float groudtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v0<blockSize><<<Grid,Block>>>(d_a, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        //for(int i = 0; i < GridSize;i++){
            //printf("res per block : %lf ",out[i]);
        //}
        //printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_v0 latency = %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}
